#include <hip/hip_runtime.h>
#include <stdio.h>
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

void initialInt(int *ip, int size)
{
    for(int i=0; i<size; i++)
    {
        ip[i]=i;
    }
}

void printMatrix(int *C, const int nx, const int ny)
{
    int *ic= C;
    printf("\nMatrix: (%d.%d)\n",nx,ny);
    for(int iy=0;iy<ny; iy++)
    {
        for(int ix = 0; ix<nx; ix++)
        {
            printf("%3d", ic[ix]);
        }
        ic += nx;
        printf("\n");
    }
    printf("\n");
}

__global__ void printThreadIdx(int *A, const int nx, const int ny) {
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    unsigned int idx = iy*nx +ix;
    printf("thread_id (%d, %d) block_id (%d, %d) coordinate (%d, %d)"
            "global index %2d ival %2d\n",threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix,iy, idx, A[idx]);
}
int main(int argc, char **argv){
    printf("%s Strating .... \n", argv[0]);
    int dev =0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d :%s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
    
    int nx = 8;
    int ny = 6;
    int nxy = nx*ny;
    int nBytes = nxy * sizeof(float);

    int *h_A;
    h_A = (int *)malloc(nBytes);
    initialInt(h_A, nx);
    printMatrix(h_A, nx, ny);

    int *d_MatA;
    hipMalloc((void **)&d_MatA, nBytes);

    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);

    dim3 block(4,2);
    dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

    printThreadIdx<<<grid, block>>>(d_MatA, nx, ny);
    hipDeviceSynchronize();

    hipFree(d_MatA);
    free(h_A);

    hipDeviceReset();

    return (0);
}