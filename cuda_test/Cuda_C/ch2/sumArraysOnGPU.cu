#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    bool match = 1;
    for(int i=0; i<N; i++){
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("Arrays do not match ! \n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if (match) printf("Arrays match.\n \n ");
}

void initialData(float *ip, int size){
    time_t t;
    srand((unsigned) time(&t));

    for(int i =0; i<size;i++) {
        ip[i] = (float)(rand() &0xFF) /10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for(int idx=0; idx<N; idx++)
    {
        C[idx] = A[idx] +B[idx];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
    int i=threadIdx.x;
    C[i]= A[i]+B[i];
}

int main(int argc, char **argv){
    printf("%s Starting ....\n", argv[0]);
    int dev = 0;
    hipSetDevice(dev);

    int nElem = 32;
    printf("Vector size %d\n", nElem);
    //malloc host memory
    size_t nBytes = nElem *sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef =(float *)malloc(nBytes);

    //initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    float *d_A, *d_B, *d_C;
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyDeviceToHost);

    dim3 block(nElem);
    dim3 grid(nElem/block.x);
    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d, %d>>>\n",grid.x, block.x);

    hipMemcpy(gpuRef,d_C, nBytes, hipMemcpyDeviceToHost);

    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipFree(h_A);
    hipFree(h_B);
    hipFree(hostRef);
    hipFree(gpuRef);

    return (0);

}
