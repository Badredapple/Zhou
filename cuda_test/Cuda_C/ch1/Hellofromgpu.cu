
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void hellofromgpu(void)
{
    printf("Hello from gpu \n");
}
int main(void)
{
    printf("Hello from cpu \n");
    hellofromgpu <<<1, 5>>>();
    hipDeviceReset();
    return 0;
}
