
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 20
__global__ void vecadd(int *a, int *b, int *c)
{
    int tid =threadIdx.x;
    if(tid < N)
    {
        c[tid] = a[tid]+b[tid];    
    }
}

int main(void)
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    for(int i=0; i<N;++i)
    {
        a[i] = -i;
        b[i] = i*i;
    }

    hipMalloc( (void **)&dev_a, N*sizeof(int) );
    hipMalloc( (void **)&dev_b, N*sizeof(int) );
    hipMalloc( (void **)&dev_c, N*sizeof(int) );
    

    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
    vecadd<<<1, N>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c,dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<N;++i)
    {
        printf("%d +%d = %d\n", a[i], b[i], c[i]);
    }


    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
