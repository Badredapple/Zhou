
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10
void vecadd(int *a, int *b, int *c)
{
    int tid =0;
    while(tid<N)
    {
        c[tid] =a[tid]+b[tid];
        ++tid;
    }
}

int main(void)
{
    int a[N], b[N], c[N];
    for(int i = 0; i<N; ++i)
    {
        a[i] = -i;
        b[i] = i*i;
    }
    vecadd (a, b,c);
        for(int i = 0; i< N;++i)
        {
            printf("%d + %d = %d \n",a[i],b[i],c[i]);
        }
    
        return 0;
}
